#include "hip/hip_runtime.h"
#include <hip/hip_runtime_api.h>

#define UMUL(a, b) ( (a) * (b) )
#define UMAD(a, b, c) ( UMUL((a), (b)) + (c) )

typedef unsigned int uint;
typedef unsigned short ushort;
typedef unsigned char uchar;


#define SHARED_MEMORY_SIZE 49152
#define MERGE_THREADBLOCK_SIZE 128

/*
 *	Function that maps value to bin in range 0 inclusive to binCOunt exclusive
 */
inline __device__ uint binOfValue(uint value, uint binCount)
{
	//TODO get some sensible function to assign bin
	return value % binCount;
	//return 0;
}

__global__ void clearHistogram(uint *d_Histogram, uint binCount)
{
	//clear histogram
	for (uint bin = UMAD(blockIdx.x, blockDim.x, threadIdx.x); bin < binCount; bin += UMUL(blockDim.x, gridDim.x))
		d_Histogram[bin] = 0;
}

//1 byte per bin kernel
__global__ void byteHistogramKernel(uint *d_PartialHistograms, uint *d_Data, uint dataCount, uint binCount)
{
	//TODO move constants out of kernel
	uint tid = UMAD(blockIdx.x, blockDim.x, threadIdx.x);
	uint threadCount = UMUL(blockDim.x, gridDim.x);
	uint binsPerThread = binCount / blockDim.x;
	
	//TODO try to limit bank conflicts
	extern __shared__ uchar s_byteHistogram[];
	//__shared__ uchar s_byteHistogram[SHARED_MEMORY_SIZE];

	//clear shared memory histogram bins assigned to this thread
	#pragma unroll
	for (uint bin = binsPerThread * threadIdx.x; bin < binsPerThread * (threadIdx.x + 1) && bin < binCount; bin++)
		s_byteHistogram[bin] = 0;
	__syncthreads();
	
	for (uint data = tid; data < dataCount; data += threadCount)//approximate
	//for (uint data = 0; data < dataCount; data++)//with do over
	{
		uint bin = binOfValue(d_Data[data], binCount);
		if(bin >= binsPerThread * threadIdx.x && bin < binsPerThread * (threadIdx.x + 1))
		{
			//update bin (no need for synchronization, only this thread can modify this bin)
			s_byteHistogram[bin]++;
			//if overflow copy to global memory
			if(s_byteHistogram[bin] == 255)
			{
				d_PartialHistograms[blockIdx.x * binCount + bin] += s_byteHistogram[bin];
				s_byteHistogram[bin] = 0;
			}
		}
		else
		{
			//disregard data has to be processed by other thread
		}
	}
	
	//copy final histogram bins assigned to this thread to global
	#pragma unroll
	for (uint bin = binsPerThread * threadIdx.x; bin < binsPerThread * (threadIdx.x + 1) && bin < binCount; bin++)
		d_PartialHistograms[blockIdx.x * binCount + bin] += s_byteHistogram[bin];
}

__global__ void shortHistogramKernel(uint *d_PartialHistograms, uint *d_Data, uint dataCount, uint binCount)
{
	//TODO move constants out of kernel
	uint tid = UMAD(blockIdx.x, blockDim.x, threadIdx.x);
	uint threadCount = UMUL(blockDim.x, gridDim.x);
	uint binsPerThread = binCount / blockDim.x;
	
	//TODO try to limit bank conflicts
	extern __shared__ ushort s_shortHistogram[];
	//__shared__ ushort s_shortHistogram[SHARED_MEMORY_SIZE/2];

	//clear shared memory histogram bins assigned to this thread
	#pragma unroll
	for (uint bin = binsPerThread * threadIdx.x; bin < binsPerThread * (threadIdx.x + 1) && bin < binCount; bin++)
		s_shortHistogram[bin] = 0;
	__syncthreads();
	
	for (uint data = tid; data < dataCount; data += threadCount)//approximate
	//for (uint data = 0; data < dataCount; data++)//with do over
	{
		uint bin = binOfValue(d_Data[data], binCount);
		if(bin >= binsPerThread * threadIdx.x && bin < binsPerThread * (threadIdx.x + 1))
		{
			//update bin (no need for synchronization, only this thread can modify this bin)
			s_shortHistogram[bin]++;
			//if overflow copy to global memory
			if(s_shortHistogram[bin] == 65535)
			{
				d_PartialHistograms[blockIdx.x * binCount + bin] += s_shortHistogram[bin];
				s_shortHistogram[bin] = 0;
			}
		}
		else
		{
			//disregard data has to be processed by other thread
		}
	}
	
	//copy final histogram bins assigned to this thread to global
	#pragma unroll
	for (uint bin = binsPerThread * threadIdx.x; bin < binsPerThread * (threadIdx.x + 1) && bin < binCount; bin++)
		d_PartialHistograms[blockIdx.x * binCount + bin] += s_shortHistogram[bin];
}

__global__ void intHistogramKernel(uint *d_PartialHistograms, uint *d_Data, uint dataCount, uint binCount)
{
	//TODO move constants out of kernel
	uint tid = UMAD(blockIdx.x, blockDim.x, threadIdx.x);
	uint threadCount = UMUL(blockDim.x, gridDim.x);
	uint binsPerThread = binCount / blockDim.x;
	
	//TODO try to limit bank conflicts
	extern __shared__ uint s_Histogram[];
	//__shared__ uint s_Histogram[SHARED_MEMORY_SIZE/4];

	//clear shared memory histogram bins assigned to this thread
	#pragma unroll
	for (uint bin = binsPerThread * threadIdx.x; bin < binsPerThread * (threadIdx.x + 1) && bin < binCount; bin++)
		s_Histogram[bin] = 0;
	__syncthreads();
	
	for (uint data = tid; data < dataCount; data += threadCount)//approximate
	//for (uint data = 0; data < dataCount; data++)//with do over
	{
		uint bin = binOfValue(d_Data[data], binCount);
		if(bin >= binsPerThread * threadIdx.x && bin < binsPerThread * (threadIdx.x + 1))
		{
			//update bin (no need for synchronization, only this thread can modify this bin)
			s_Histogram[bin]++;
		}
		else
		{
			//disregard data has to be processed by other thread
		}
	}
	
	//copy final histogram bins assigned to this thread to global
	#pragma unroll
	for (uint bin = binsPerThread * threadIdx.x; bin < binsPerThread * (threadIdx.x + 1) && bin < binCount; bin++)
		d_PartialHistograms[blockIdx.x * binCount + bin] += s_Histogram[bin];
}


__global__ void mergePartialHistogramsKernel(uint *d_Histogram, uint *d_PartialHistograms, uint histogramCount,	uint binCount)
{
	for (uint bin = blockIdx.x; bin < binCount; bin += gridDim.x)
	{
		uint sum = 0;
		for (uint histogramIndex = threadIdx.x; histogramIndex < histogramCount; histogramIndex += MERGE_THREADBLOCK_SIZE)
		{
			sum += d_PartialHistograms[bin + histogramIndex * binCount];
		}
	
		__shared__ uint data[MERGE_THREADBLOCK_SIZE];
		data[threadIdx.x] = sum;
	
		for (uint stride = MERGE_THREADBLOCK_SIZE / 2; stride > 0; stride >>= 1)
		{
			__syncthreads();
	
			if (threadIdx.x < stride)
			{
				data[threadIdx.x] += data[threadIdx.x + stride];
			}
		}
		
		if (threadIdx.x == 0)
		{
			d_Histogram[bin] = data[0];
		}
	}
}

static uint *d_PartialHistograms;

extern "C" void initPartialHistograms(uint partialHistogramCount, uint binCount)
{
    checkCudaErrors(hipMalloc((void **)&d_PartialHistograms, partialHistogramCount * binCount * sizeof(uint)));
}

//Internal memory deallocation
extern "C" void closePartialHistograms(void)
{
    checkCudaErrors(hipFree(d_PartialHistograms));
}

extern "C" void approxHistogramGPU(uint *d_Histogram, void *d_Data, uint byteCount, uint binCount, hipDeviceProp_t deviceProp)
{
	uint partialHistogramCount = 128;
	initPartialHistograms(partialHistogramCount, binCount);
	
	clearHistogram<<<partialHistogramCount, 512>>>(d_Histogram, binCount);
	getLastCudaError("clearHistogram() execution failed\n");
	clearHistogram<<<partialHistogramCount, 512>>>(d_PartialHistograms, partialHistogramCount * binCount);
	getLastCudaError("clearHistogram() execution failed\n");
	
	//dynamically get shared memory size from device
	//dynamically get bytes per bin
	uint bytesPerBin = SHARED_MEMORY_SIZE / binCount;
	
	if(bytesPerBin == 0)
	{
		// Too many bins. Cannot be processed on given hardware
		printf("... execution failed too many bins\n");
	}
	else if (bytesPerBin == 1)
	{
		printf("... using byteHistogramKernel\n");
		//use kernel with 1 byte per bin
		byteHistogramKernel<<<partialHistogramCount, 256, binCount * sizeof(uchar) >>>(d_PartialHistograms, (uint *) d_Data, byteCount / sizeof(uint), binCount);
		hipDeviceSynchronize();
		getLastCudaError("byteHistogramKernel() execution failed\n");

		mergePartialHistogramsKernel<<<256, MERGE_THREADBLOCK_SIZE>>>(d_Histogram, d_PartialHistograms, partialHistogramCount, binCount);
		hipDeviceSynchronize();
		getLastCudaError("mergePartialHistogramsKernel() execution failed\n");
	}
	else if (bytesPerBin == 2 || bytesPerBin == 3)
	{
		printf("... using shortHistogramKernel\n");
		//use kernel with 2 byte per bin
		shortHistogramKernel<<<partialHistogramCount, 256, binCount * sizeof(ushort) >>>(d_PartialHistograms, (uint *) d_Data, byteCount / sizeof(uint), binCount);
		getLastCudaError("shortHistogramKernel() execution failed\n");

		mergePartialHistogramsKernel<<<256, MERGE_THREADBLOCK_SIZE>>>(d_Histogram, d_PartialHistograms, partialHistogramCount, binCount);
		getLastCudaError("mergePartialHistogramsKernel() execution failed\n");
	}
	else if (bytesPerBin > 3 )
	{
		printf("... using intHistogramKernel\n");
		//use kernel with 4 byte per bin
		intHistogramKernel<<<partialHistogramCount, 256, binCount * sizeof(uint) >>>(d_PartialHistograms, (uint *) d_Data, byteCount / sizeof(uint), binCount);
		getLastCudaError("intHistogramKernel() execution failed\n");

		mergePartialHistogramsKernel<<<256, MERGE_THREADBLOCK_SIZE>>>(d_Histogram, d_PartialHistograms, partialHistogramCount, binCount);
		getLastCudaError("mergePartialHistogramsKernel() execution failed\n");
	}
	
	closePartialHistograms();
}